
#include <hip/hip_runtime.h>
// atomicMax is not implemented for floats,
// solution copied https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ __forceinline__ float atomicMaxf(float * addr, float value) {
    if (signbit(value)) {
        return __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));        
    } else {
        return __int_as_float(atomicMax((int *)addr, __float_as_int(value)));
    }
}

__device__ unsigned int get_strided_index(
    unsigned int idx,
    size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int strided_i = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        unsigned int dim_idx = num_dims - 1 - d;
        strided_i += (idx % dims[dim_idx]) * strides[dim_idx];
        idx /= dims[dim_idx];
    }
    return strided_i;
}

__device__ unsigned int get_unstrided_index(
    const unsigned int strided_i,
    const size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int idx = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        idx *= dims[d];
        idx += strides[d] == 0 ? 0 : (strided_i / strides[d]) % dims[d];
    }
    return idx;
}

extern "C" __global__ void fill_with(float *buf, float value, const size_t numel) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }
    buf[i] = value;
}

// Sourced from https://graphics.stanford.edu/~seander/bithacks.html#RoundUpPowerOf2
__device__ __forceinline__ unsigned int next_power_of_two(unsigned int v) {
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v++;
    return v;
}

// Efficiently computes the max of each chunk in "data" of size chunk_len, and
// stores the maximums in out[i / chunk_len]
__device__ void chunk_max(
    const size_t numel,
    const size_t chunk_len,
    const float data,
    float* out
) {
    __shared__ float buf[1024];
    // assumes that threads where i >= numel have already exited
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int block_i = threadIdx.x;
    buf[block_i] = data;

    unsigned int chunk_i = i % chunk_len;
    unsigned int chunk_start = max((int)(block_i - chunk_i), 0);
    unsigned int chunk_end = min((unsigned int)(block_i + chunk_len - chunk_i), blockDim.x);

    chunk_i = block_i - chunk_start;

    size_t max_chunk_len = min(chunk_end - chunk_start, blockDim.x);
    size_t incr = next_power_of_two(max_chunk_len) >> 1;

    __syncthreads();

    // Uses sequential addressing as discussed in
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    for (; incr > 0; incr >>= 1) {
        unsigned int block_i_2 = block_i + incr;

        if (block_i_2 < chunk_end && chunk_i < incr) {
            // This is sound because __syncthreads and the conditions above
            // ensure that no data races occur
            buf[block_i] = fmaxf(buf[block_i], buf[block_i_2]);
        }

        __syncthreads();
    }

    if (block_i == chunk_start) {
        atomicMaxf(out + i / chunk_len, buf[block_i]);
    }
}

// strides and dims specify how to index inp to put all summed elements next to
// each other, and chunk_len is len(inp) / len(out)
extern "C" __global__ void max_to_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t chunk_len,
    const float *inp,
    const size_t *dims,
    const size_t *strides,
    float *out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numel) {
        return;
    }

    unsigned int inp_i = get_strided_index(i, num_dims, dims, strides);
    chunk_max(numel, chunk_len, inp[inp_i], out);
}

// Accepts pre-broadcasted strides for both input & output.
// So both inp & out are expected to be broadcasted to the same size.
extern "C" __global__ void max_to_backward(
    const size_t numel,
    const size_t num_dims,
    const float elems_per_thread,
    const size_t *dims,
    const float *inp,
    float *grad_inp,
    const size_t *inp_strides,
    const float *out,
    const float *grad_out,
    const size_t *out_strides
) {
    unsigned int inp_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (inp_i >= numel) {
        return;
    }

    unsigned int i = get_unstrided_index(inp_i, num_dims, dims, inp_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    auto tmp = inp[inp_i] == out[out_i] ? grad_out[out_i] : 0.0;
    grad_inp[inp_i] += tmp * elems_per_thread;
}
